#include "hip/hip_runtime.h"
#include <assert.h>
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

/* Template project which demonstrates the basics on how to setup a project 
* example application.
* Host code.
*/
/*********************************************/
// datarr columns:  
// datarr[][][0][] is Vm       (mV)
// datarr[][][1][] is dVmdt    (mV/msec)
// datarr[][][2][] is IK1      (uA/cm^2)
// datarr[][][3][] is Ix1      (uA/cm^2)
// datarr[][][4][] is x1       (unitless)
// datarr[][][5][] is INa      (uA/cm^2)
// datarr[][][6][] is m        (unitless)
// datarr[][][7][] is h        (unitless)
// datarr[][][8][] is Is       (uA/cm^2)
// datarr[][][9][] is d       (unitless)
// datarr[][][10][] is f       (unitless)
// datarr[][][11][] is Cai     (mole/L)
// datarr[][][12][] is Isum    (uA/cm^2)
// datarr[][][13][] is Diff    (mV/msec) 
// datarr[][][14][] is Istim   (uA/cm^2)  Istim should always be the last variable in datarr
/*********************************************/
// derivarr columns: 
// derivarr[0] is current time  (msec)
// derivarr[1] is dx1dt         (unitless)
// derivarr[2] is dmdt          (unitless)
// derivarr[3] is dhdt          (unitless)
// derivarr[4] is dddt          (unitless)
// derivarr[5] is dfdt          (unitless)
// derivarr[6] is dCaidt        (mole/L)
/*********************************************/
// Constants: 
// constarr[0] is gK1   (mmho/cm^2)
// constarr[1] is gNa   (mmho/cm^2)
// constarr[2] is ENa   (mV)  
// constarr[3] is gx1   (mmho/cm^2)
// constarr[4] is gs    (mmho/cm^2)
// constarr[5] is Cm    (uF/cm^2)
// constarr[6] is kCa   (msec^-1)
// constarr[7] is gNaC  (mmho/cm^2)     /* should be set to zero in brdr2dtask.dat */
// constarr[8] is Dpara   (cm^2/msec)
// constarr[9] is Dperpen (cm^2/msec)
// constarr[10] is theta  (degrees)
// constarr[11] is sigma  (unitless)
// constarr[12] is A      (unitless)
/*********************************************/
// Diffusion Tensor:  note-> D12=D21
// D[0][0] is D11     (cm^2/msec)
// D[0][1] is D12     (cm^2/msec)
// D[1][0] is D21     (cm^2/msec)
// D[1][1] is D22     (cm^2/msec)
/*********************************************/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
//#include <sys/dir.h>  use to check for and/or create data directory?
#include "brdr2d.h"
#include "brdr2dinout.c"
//#include "brdr2dequations.c"

// includes, project
#include <cutil_inline.h>
#include "cuPrintf.cu"
// includes, kernels
#include <brdr2d_kernel.cu>


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void invokeGPU(int argc, char** argv);
void GPU_Mem_init(void);

extern "C"
void computeGold( float* reference, float* idata, const unsigned int len);
void initialize();
void openfiles();
void buildedgestim();
void buildptstim();
void build2ptstims();
void buildbarstim1();
void buildbarstim2();
void buildcrossstim();
void stimulate();
void blockonoff();
void brgates();
void brcurrents();
void bcs();
void output();
void closefiles();
void brfc();
void readbrfc();
void stability();
double rtclock();

const int VAR_N = 1;
int	size_int = sizeof(int) * VAR_N;
int size_double = sizeof(double) * VAR_N;


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
	printf("Initializing ... \n");
	initialize();
	if (stimnum > 0)
	{
		printf("Building stimulus matrix ... \n");
		buildptstim();  
	}
	printf("Opening files ... \n");
	openfiles();

	step=0;		/* output function calls step */
  	printf("Writing initial conditions ... \n");
 	output();


	time_t ttime=time(0);            // Get current time
  	char *stime=ctime(&ttime);
  	printf("%s",stime);
	
	invokeGPU(argc, argv);

	printf("Saving final conditions...\n\n");

	brfc();
 	printf("         tfinal: %5.3f msec\n",tfinal);
  	printf("     Final time: %5.3f msec\n",derivarr[0]);
  	printf("         Nsteps: %10.2f\n",Nsteps);
  	printf("Number of steps: %d\n",step);
  	printf("             Nx: %d\n",Nx);
  	printf("             Ny: %d\n",Ny);

  	ttime=time(0);                 // Get current time
  	stime=ctime(&ttime);
  	printf("%s\n",stime);
    
  	closefiles();
}

void
invokeGPU(int argc, char *argv[])
{
	int i,j,k;

	//allocate GPU memory, copy data from host to device	
	GPU_Mem_init();
	
	//unsigned int timer = 0;
  	printf("Entering time loop ... \n");
	//cutilCheckError( cutCreateTimer( &timer));
	//cutilCheckError( cutStartTimer( timer));
	step = 1;
	derivarr[0] += dt;
	deriv3darr[0][0][0] += dt;  // update time (msec) 
	cutilSafeCall(hipMemcpy(d_derivarr,
				deriv3darr[0][0],
				sizeof(double), hipMemcpyHostToDevice));			

	// setup execution parameters
	THREAD_DIMX = atoi(argv[1]);
	THREAD_DIMY = atoi(argv[2]);
	if(Nx%THREAD_DIMX != 0){
		printf("Nx is %d, Thread_Dimx is %d, Nx % Thread_Dimx != 0 return\n",Nx,THREAD_DIMX); 
		return;
	}
	else BLOCK_DIMX = Nx/THREAD_DIMX;
	if(Ny%THREAD_DIMY != 0){
		printf("Ny is %d, Thread_Dimy is %d, Ny % Thread_Dimy != 0 return\n",Ny,THREAD_DIMY);
		return;
	}
	else BLOCK_DIMY = Ny/THREAD_DIMY;
	dim3 dimGrid(BLOCK_DIMX,BLOCK_DIMY,1);
	dim3 dimBlock(THREAD_DIMX,THREAD_DIMY,1);

	cudaPrintfInit();
	
	double gpu_start = rtclock();
	double stim_time=0;
	double block_time=0;
	double cur_time=0;
	double gate_time=0;
	double bcs_time=0;
	double mem_time=0;
	double time_temp;
	double update_time=0;
	while (derivarr[0] <= tfinal+dt && step <= Nsteps + 1)
	{
		// from (1 to Nx) instead of (0 to Nx+1)
		// do not loop through ghost points */
		//GPU Kernel Execution
		time_temp = rtclock();
		if(stimnum>0) d_stimulate_kernel<<<dimGrid,dimBlock>>>(stimnum,d_datarr,d_stimarr,d_derivarr,varnum,step,Istimamp,Nx,Ny,stimint);
		hipDeviceSynchronize();
		stim_time += (double)(rtclock()-time_temp);

		time_temp = rtclock();
		if(blocktimenum>0) d_blockonoff_kernel<<<dimGrid,dimBlock>>>(blocktimenum, d_derivarr, d_blocktimes, d_block, Nx, Ny);
		hipDeviceSynchronize();
		block_time += (double)(rtclock()-time_temp);

		time_temp = rtclock();
		d_brgates_kernel<<<dimGrid,dimBlock>>>(varnum, d_datarr, d_derivarr, d_constarr, step, Nx, Ny);
		hipDeviceSynchronize();
		gate_time += (double)(rtclock()-time_temp);

		time_temp = rtclock();
		d_brcurrents_kernel<<<dimGrid,dimBlock>>>(stimnum, d_datarr, d_derivarr, step, Istimamp, Nx,Ny, varnum, d_constarr, d_Afield, d_block, d_Dp, dt);
		hipDeviceSynchronize();
		cur_time += (double)(rtclock()-time_temp);

		time_temp = rtclock();
		dim3 dimGrid1(1,1,1);
    	dim3 dimBlock1(1,1,1);
		kernel_call_device_bcs<<< dimGrid1, dimBlock1 >>>(dx, dy, d_D, BC, step, Nx, Ny, varnum, d_Dp, d_datarr, d_derivarr, dt);  
		hipDeviceSynchronize();		
		cutilCheckMsg("CUDA Kernel");
		bcs_time += (double)(rtclock()-time_temp);
		
		time_temp = rtclock();
		NinePointLaplacian<<< dimGrid, dimBlock >>>(step, varnum, Nx, Ny, Dp[0][0],Dp[0][1],Dp[1][0],Dp[1][1], d_datarr);  
		hipDeviceSynchronize();		
		cutilCheckMsg("Laplacian CUDA Kernel");
		update_time += (double)(rtclock()-time_temp);
		
		time_temp = rtclock();
		if (step % rpN == 0) {
			// Coalescing hipMemcpy
			cutilSafeCall(hipMemcpy(linear_datarr, 
				             d_datarr, 
				             (Nx+2)*(Ny+2)*varnum*2*sizeof(double),
				             hipMemcpyDeviceToHost));
		    
			// copy host memory from device
			for (int l = 0; l < 2; l++)
			{
				for (k = 0; k < varnum; k++)
				{
					for (i = 0; i < (Nx+2); i++)
					{
						for (j = 0; j < (Ny+2); j++)
						{
							datarr[l][k][i][j] = 
						     	*(linear_datarr+
							l*(Nx+2)*(Ny+2)*varnum+
							k*(Nx+2)*(Ny+2)+
							i*(Ny+2)+
							j);
						}
					}
				}
			}
		 
		       output();       
	  
			printf("%4.4e msec, Vm(%d,%d): %3.2f mV GPU\n",
				derivarr[0], mNx, mNy, datarr[step%2][0][mNx][mNy]);
		}
		mem_time += (double)(rtclock()-time_temp);
		step++;
		
		derivarr[0] += dt;
		deriv3darr[0][0][0] += dt;  // update time (msec) 
     	
	}
	double gpu_end = rtclock();    
	
	printf("total         time is %.2lf\n",(double)(gpu_end-gpu_start));	
	printf("Kernel stim   time is %.2lf\n",stim_time);
	printf("Kernel block  time is %.2lf\n",block_time);
	printf("Kernel gate   time is %.2lf\n",gate_time);
	printf("Kernel cur    time is %.2lf\n",cur_time);
	printf("Kernel bcs    time is %.2lf\n",bcs_time);
	printf("Kernel update time is %.2lf\n",update_time);
	printf("memory copy   time is %.2lf\n",mem_time);
	printf("GPU           time is %.2lf\n",stim_time+block_time+gate_time+cur_time+bcs_time+update_time);				   
	cudaPrintfEnd();
	// check if kernel execution generated and error
	cutilCheckMsg("Kernel execution failed");
	
	//cutilCheckError( cutStopTimer( timer));
	//printf( "Processing time: %f (ms)\n", cutGetTimerValue( timer));
	//cutilCheckError( cutDeleteTimer( timer));

	// cleanup memory
	hipDeviceReset();
}

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

void buildptstim(void){
/* point stimulus  */
  int i,j,k;
  int Nxx,Nyy;
  double stimsizeir;
  double radius;

  Nxx=(unsigned int)(floor(Nx/2));
  Nyy=(unsigned int)(floor(Ny/2));
  printf("Point stimulus centered at %d,%d\n",Nxx,Nyy);
  stimsizeir=floor(stimsize1/dx);
  printf("Point stimulus radius: %4.3f cm, %4.3f pixels\n",stimsize1,stimsizeir);
  for (k=0;k<stimnum;++k){
    for (i=0;i<=Nx;++i){
      for (j=0;j<=Ny;++j){
        radius=sqrt(((double)(Nxx-i))*((double)(Nxx-i)) + ((double)(Nyy-j))*((double)(Nyy-j)));
    if (radius<=stimsizeir){
          stimarr[i][j][k][0]=stimes[k];
    }
      }
    }
  }
}

void blockonoff(void)
{
  int i,m,n;

  for (i=0;i<blocktimenum;++i){
    if ((derivarr[0]>=blocktimes[i][0])&&(blocktimes[i][1]==0.0)){
      printf("Changing block conditions: %4.3f msec \n",derivarr[0]);
      blocktimes[i][1]=1.0;
      for (m=1;m<Nx+1;++m){
        for (n=1;n<Ny+1;++n){
      if (block[m][n]==0) block[m][n]=1;
    }
      }
    }
  }

}

void GPU_Mem_init(void)
{
	int i, j, k;		// loop index
	long int xyzw_size;
	long int xyzw_stim_size;
	long int xyz_deriv_size;
	

	
	// Use device with highest Gflops/s
	hipSetDevice( cutGetMaxGflopsDeviceId() );
	

	

	int* d_varnum; 
	hipMalloc((void **)&d_varnum, size_int);
	hipMemcpy(d_varnum, &varnum, size_int, hipMemcpyHostToDevice);

	int* d_step;    
	hipMalloc((void **)&d_step, size_int);
	hipMemcpy(d_step, &step, size_int, hipMemcpyHostToDevice);

	double* d_Istimamp;
	hipMalloc((void **)&d_Istimamp, size_double);
	hipMemcpy(d_Istimamp, &Istimamp, size_double, hipMemcpyHostToDevice);

	int* d_Nx;             
	hipMalloc((void **)&d_Nx, size_int);
	hipMemcpy(d_Nx, &Nx, size_int, hipMemcpyHostToDevice);

	int* d_Ny;             
	hipMalloc((void **)&d_Ny, size_int);
	hipMemcpy(d_Ny, &Ny, size_int, hipMemcpyHostToDevice);

	int* d_blocktimenum;
	hipMalloc((void **)&d_blocktimenum, size_int);
	hipMemcpy(d_blocktimenum, &blocktimenum, size_int, hipMemcpyHostToDevice);

	double* d_stimint;             
	hipMalloc((void **)&d_stimint, size_double);
	hipMemcpy(d_stimint, &stimint, size_double, hipMemcpyHostToDevice);

	double* d_dt;             
	hipMalloc((void **)&d_dt, size_double);
	hipMemcpy(d_dt, &dt, size_double, hipMemcpyHostToDevice);

	int* d_BC; 
	hipMalloc((void **)&d_BC, size_int);
	hipMemcpy(d_BC, &BC, size_int, hipMemcpyHostToDevice);
	
	double* d_dx;
	hipMalloc((void **)&d_dx, size_double);
	hipMemcpy(d_dx, &dx, size_double, hipMemcpyHostToDevice);

	double* d_dy;
	hipMalloc((void **)&d_dy, size_double);
	hipMemcpy(d_dy, &dy, size_double, hipMemcpyHostToDevice);

	xyzw_size = (Nx+2) * (Ny+2) * (varnum) * (datarr4dim) * sizeof(double);
	xyzw_stim_size = Nx * Ny * stimnum * 2 * sizeof(double);
	xyz_deriv_size = Nx * Ny * derivnum * sizeof(double);
	
	// allocate host memory
	// should have already allocated host memory 

 	// allocate device memory
	cutilSafeCall(hipMalloc((void**)&d_datarr, 
			xyzw_size));	
	cutilSafeCall(hipMalloc((void**)&d_stimarr, 
			xyzw_stim_size));	
	cutilSafeCall(hipMalloc((void**)&d_derivarr, 
			xyz_deriv_size));
	/* d_blocktimes */
	cutilSafeCall(hipMalloc((void**)&d_blocktimes,
			blocktimenum*2*sizeof(double)));
	/* d_block */
	cutilSafeCall(hipMalloc((void**)&d_block,
			(Nx+2)*(Ny+2)*sizeof(int)));
	/* d_constarr */
	cutilSafeCall(hipMalloc((void**)&d_constarr,
			constnum*sizeof(double)));
	/* d_Afield */
	cutilSafeCall(hipMalloc((void**)&d_Afield,
			Nx*Ny*sizeof(double)));
	/* d_Dp */
	cutilSafeCall(hipMalloc((void**)&d_Dp,
			2*2*sizeof(double)));
	/* d_D */
	cutilSafeCall(hipMalloc((void**)&d_D,
			2*2*sizeof(double)));
    		
	
	linear_datarr = (double *) malloc ( (unsigned int)
             (sizeof(double)*2*varnum*(Ny+2)*(Nx+2)));

	if (NULL == linear_datarr)
	{
		printf("Malloc Failed\n");
		exit(-1);
	}
                                    

    // copy host memory to device
       	for (int l = 0; l < 2; l++)
	{
		for (k = 0; k < varnum; k++)
		{
			for (i = 0; i < (Nx+2); i++)
			{
				for (j = 0; j < (Ny+2); j++)
                		{
		             		*(linear_datarr+
		                	l*(Ny+2)*(Nx+2)*varnum+
		                	k*(Ny+2)*(Nx+2)+
		                	i*(Ny+2)+
		                	j) = datarr[l][k][i][j]; 
                		}
			}
		}
	}
 
	// Coalescing hipMemcpy
	cutilSafeCall(hipMemcpy(d_datarr, 
                             linear_datarr, 
                             (Nx+2)*(Ny+2)*varnum*2*sizeof(double),
                             hipMemcpyHostToDevice));


    
	linear_stimarr = (double*)malloc((unsigned int)
                        (Nx*Ny*stimnum*2*sizeof(double))); 
	if (NULL == linear_stimarr)
	{
		printf("Malloc Linear Stimarr Failed\n");
		exit(-1);
	}
	//stim array
	for (i = 0; i < Nx; ++i)
	{
		for (j = 0; j < Ny; ++j)
		{
			for (k = 0; k < stimnum; ++k)
			{
                		for (int l = 0; l < 2; ++l)
				{
				     *(linear_stimarr+
				        i*Ny*stimnum*2+
				        j*stimnum*2+
				        k*2+
				        l) = stimarr[i][j][k][l]; 
				}
			}
		}
	} 

	cutilSafeCall( hipMemcpy(
		            d_stimarr,
            		linear_stimarr, 
		            Nx*Ny*stimnum*2*sizeof(double), 
                    hipMemcpyHostToDevice) );


    
	linear_deriv3darr = (double*)malloc((unsigned int)
                        (Nx*Ny*derivnum*sizeof(double))); 
	if (NULL == linear_deriv3darr)
	{
		printf("Malloc Linear Deriv3darr Failed\n");
		exit(-1);
	}
	//derive3d array
	for (i = 0; i < Nx; ++i)
	{
		for (j = 0; j < Ny; ++j)
		{
			for (k = 0; k < derivnum; ++k)
			{
		             *(linear_deriv3darr+
		                i*Ny*derivnum+
		                j*derivnum+
		                k) = deriv3darr[i][j][k]; 
			}
		}
	} 

	cutilSafeCall( hipMemcpy(
		            d_derivarr,
            		linear_deriv3darr, 
		            Nx*Ny*derivnum*sizeof(double), 
                    hipMemcpyHostToDevice) );
	
	/* 1D array just Memcpy */
	/* d_blocktimes */
	for (i = 0; i < blocktimenum; i++)
	{
		cutilSafeCall(hipMemcpy(
					d_blocktimes+i*2,
					blocktimes[i], 
					2*sizeof(double), hipMemcpyHostToDevice) );
	}	
   	/* d_block */
	for (i = 0; i < Nx + 2; i++)
	{	
		cutilSafeCall(hipMemcpy(
					d_block+i*(Ny+2),
					block[i], 
					(Ny+2)*sizeof(int), hipMemcpyHostToDevice) );
	}	
	/* d_constarr */
	cutilSafeCall(hipMemcpy(d_constarr, constarr, constnum*sizeof(double),
			hipMemcpyHostToDevice));
   	/* d_Afield */
	for (i = 0; i < Nx; i++)
	{	
		cutilSafeCall(hipMemcpy(
					d_Afield+i*Ny,
					Afield[i], 
					Ny*sizeof(double), hipMemcpyHostToDevice) );
	}
 	/* d_Dp */		
	for (i = 0; i < 2; i++)
	{	
		cutilSafeCall(hipMemcpy(
					d_Dp+i*2,
					Dp[i], 
					2*sizeof(double), hipMemcpyHostToDevice) );
	}
	/* d_D */
	for (i = 0; i < 2; i++)
	{	
		cutilSafeCall(hipMemcpy(
					d_D+i*2,
					D[i], 
					2*sizeof(double), hipMemcpyHostToDevice) );
	}
	return;
}
